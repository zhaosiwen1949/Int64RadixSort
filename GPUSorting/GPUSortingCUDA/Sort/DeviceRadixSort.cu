#include "hip/hip_runtime.h"
/******************************************************************************
 * GPUSorting
 * Device Level 8-bit LSD Radix Sort using reduce then scan
 *
 * SPDX-License-Identifier: MIT
 * Author:  Thomas Smith 2/21/2024
 * https://github.com/b0nes164/GPUSorting
 *
 ******************************************************************************/
#include "DeviceRadixSort.cuh"

#define RADIX               256     //Number of digit bins
#define RADIX_MASK          255     //Mask of digit bins, to extract digits
#define RADIX_LOG           8       //log2(RADIX)

#define SEC_RADIX_START     256     //Offset for retrieving value from global histogram buffer
#define THIRD_RADIX_START   512     //Offset for retrieving value from global histogram buffer
#define FOURTH_RADIX_START  768     //Offset for retrieving value from global histogram buffer

//For the upfront global histogram kernel
#define PART_SIZE			7680
#define VEC_PART_SIZE		1920

//For the digit binning
#define BIN_PART_SIZE       7680                                    //Partition tile size in k_DigitBinning
#define BIN_HISTS_SIZE      4096                                    //Total size of warp histograms in shared memory in k_DigitBinning
#define BIN_SUB_PART_SIZE   480                                     //Subpartition tile size of a single warp in k_DigitBinning
#define BIN_WARPS           16                                      //Warps per threadblock in k_DigitBinning
#define BIN_KEYS_PER_THREAD 15                                      //Keys per thread in k_DigitBinning
#define BIN_SUB_PART_START  (WARP_INDEX * BIN_SUB_PART_SIZE)        //Starting offset of a subpartition tile
#define BIN_PART_START      (blockIdx.x * BIN_PART_SIZE)			//Starting offset of a partition tile

//for the chained scan with decoupled lookback
#define FLAG_NOT_READY      0                                       //Flag value inidicating neither inclusive sum, nor reduction of a partition tile is ready
#define FLAG_REDUCTION      1                                       //Flag value indicating reduction of a partition tile is ready
#define FLAG_INCLUSIVE      2                                       //Flag value indicating inclusive sum of a partition tile is ready
#define FLAG_MASK           3                                       //Mask used to retrieve flag values

__global__ void DeviceRadixSort::Upsweep(
    uint32_t* sort,
    uint32_t* globalHist,
    uint32_t* passHist,
    uint32_t size,
    uint32_t radixShift)
{
    __shared__ uint32_t s_globalHist[RADIX * 2];

    //clear shared memory
    for (uint32_t i = threadIdx.x; i < RADIX * 2; i += blockDim.x)
        s_globalHist[i] = 0;
    __syncthreads();
    
    //histogram
    {
        //64 threads : 1 histogram in shared memory
        uint32_t* s_wavesHist = &s_globalHist[threadIdx.x / 64 * RADIX];

        if (blockIdx.x < gridDim.x - 1)
        {
            const uint32_t partEnd = (blockIdx.x + 1) * VEC_PART_SIZE;
            for (uint32_t i = threadIdx.x + (blockIdx.x * VEC_PART_SIZE); i < partEnd; i += blockDim.x)
            {
                const uint4 t = reinterpret_cast<uint4*>(sort)[i];
                atomicAdd(&s_wavesHist[t.x >> radixShift & RADIX_MASK], 1);
                atomicAdd(&s_wavesHist[t.y >> radixShift & RADIX_MASK], 1);
                atomicAdd(&s_wavesHist[t.z >> radixShift & RADIX_MASK], 1);
                atomicAdd(&s_wavesHist[t.w >> radixShift & RADIX_MASK], 1);
            }
        }

        if (blockIdx.x == gridDim.x - 1)
        {
            for (uint32_t i = threadIdx.x + (blockIdx.x * PART_SIZE); i < size; i += blockDim.x)
            {
                const uint32_t t = sort[i];
                atomicAdd(&s_wavesHist[t >> radixShift & RADIX_MASK], 1);
            }
        }
    }
    __syncthreads();

    //reduce to the first hist, pass out, begin prefix sum
    for (uint32_t i = threadIdx.x; i < RADIX; i += blockDim.x)
    {
        s_globalHist[i] += s_globalHist[i + RADIX];
        passHist[i * gridDim.x + blockIdx.x] = s_globalHist[i];
        s_globalHist[i] = InclusiveWarpScanCircularShift(s_globalHist[i]);
    }	
    __syncthreads();

    if (threadIdx.x < (RADIX >> LANE_LOG))
        s_globalHist[threadIdx.x << LANE_LOG] = ActiveExclusiveWarpScan(s_globalHist[threadIdx.x << LANE_LOG]);
    __syncthreads();
    
    //Atomically add to device memory
    for (uint32_t i = threadIdx.x; i < RADIX; i += blockDim.x)
        atomicAdd(&globalHist[i + (radixShift << 5)], s_globalHist[i] + (getLaneId() ? __shfl_sync(0xfffffffe, s_globalHist[i - 1], 1) : 0));
}

__global__ void DeviceRadixSort::Scan(
    uint32_t* passHist,
    uint32_t threadBlocks)
{
    __shared__ uint32_t s_scan[128];

    uint32_t reduction = 0;
    const uint32_t circularLaneShift = getLaneId() + 1 & LANE_MASK;
    const uint32_t partitionsEnd = threadBlocks / blockDim.x * blockDim.x;
    const uint32_t digitOffset = blockIdx.x * threadBlocks;

    uint32_t i = threadIdx.x;
    for (; i < partitionsEnd; i += blockDim.x)
    {
        s_scan[threadIdx.x] = passHist[i + digitOffset];
        s_scan[threadIdx.x] = InclusiveWarpScan(s_scan[threadIdx.x]);
        __syncthreads();

        if (threadIdx.x < (blockDim.x >> LANE_LOG))
        {
            s_scan[(threadIdx.x + 1 << LANE_LOG) - 1] = 
                ActiveInclusiveWarpScan(s_scan[(threadIdx.x + 1 << LANE_LOG) - 1]);
        }
        __syncthreads();

        passHist[circularLaneShift + (i & ~LANE_MASK) + digitOffset] =
            (getLaneId() != LANE_MASK ? s_scan[threadIdx.x] : 0) +
            (threadIdx.x >= LANE_COUNT ? __shfl_sync(0xffffffff, s_scan[threadIdx.x - 1], 0) : 0) +
            reduction;

        reduction += s_scan[blockDim.x - 1];
        __syncthreads();
    }

    if(i < threadBlocks)
        s_scan[threadIdx.x] = passHist[i + digitOffset];
    s_scan[threadIdx.x] = InclusiveWarpScan(s_scan[threadIdx.x]);
    __syncthreads();

    if (threadIdx.x < (blockDim.x >> LANE_LOG))
    {
        s_scan[(threadIdx.x + 1 << LANE_LOG) - 1] =
            ActiveInclusiveWarpScan(s_scan[(threadIdx.x + 1 << LANE_LOG) - 1]);
    }
    __syncthreads();

    const uint32_t index = circularLaneShift + (i & ~LANE_MASK);
    if (index < threadBlocks)
    {
        passHist[index + digitOffset] =
            (getLaneId() != LANE_MASK ? s_scan[threadIdx.x] : 0) +
            (threadIdx.x >= LANE_COUNT ?
            s_scan[(threadIdx.x & ~LANE_MASK) - 1] : 0) +
            reduction;
    }
}

__global__ void DeviceRadixSort::DownsweepKeysOnly(
    uint32_t* sort, 
    uint32_t* alt, 
    uint32_t* globalHist,
    uint32_t* passHist,
    uint32_t size, 
    uint32_t radixShift)
{
    __shared__ uint32_t s_warpHistograms[BIN_PART_SIZE];
    __shared__ uint32_t s_localHistogram[RADIX];
    volatile uint32_t* s_warpHist = &s_warpHistograms[WARP_INDEX << RADIX_LOG];

    //clear shared memory
    for (uint32_t i = threadIdx.x; i < BIN_HISTS_SIZE; i += blockDim.x)
        s_warpHistograms[i] = 0;

    //load keys
    uint32_t keys[BIN_KEYS_PER_THREAD];
    if (blockIdx.x < gridDim.x - 1)
    {
        #pragma unroll
        for (uint32_t i = 0, t = getLaneId() + BIN_SUB_PART_START + BIN_PART_START; i < BIN_KEYS_PER_THREAD; ++i, t += LANE_COUNT)
            keys[i] = sort[t];
    }

    //To handle input sizes not perfect multiples of the partition tile size,
    //load "dummy" keys, which are keys with the highest possible digit.
    //Because of the stability of the sort, these keys are guaranteed to be 
    //last when scattered. This allows for effortless divergence free sorting
    //of the final partition.
    if (blockIdx.x == gridDim.x - 1)
    {
        #pragma unroll
        for (uint32_t i = 0, t = getLaneId() + BIN_SUB_PART_START + BIN_PART_START; i < BIN_KEYS_PER_THREAD; ++i, t += LANE_COUNT)
            keys[i] = t < size ? sort[t] : 0xffffffff;
    }
    __syncthreads();

    //WLMS
    uint16_t offsets[BIN_KEYS_PER_THREAD];
    #pragma unroll
    for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
    {
        unsigned warpFlags = 0xffffffff;
        #pragma unroll
        for (int k = 0; k < RADIX_LOG; ++k)
        {
            const bool t2 = keys[i] >> k + radixShift & 1;
            warpFlags &= (t2 ? 0 : 0xffffffff) ^ __ballot_sync(0xffffffff, t2);
        }
        const uint32_t bits = __popc(warpFlags & getLaneMaskLt());
        uint32_t preIncrementVal;
        if (bits == 0)
            preIncrementVal = atomicAdd((uint32_t*)&s_warpHist[keys[i] >> radixShift & RADIX_MASK], __popc(warpFlags));

        offsets[i] = __shfl_sync(0xffffffff, preIncrementVal, __ffs(warpFlags) - 1) + bits;
    }
    __syncthreads();

    //exclusive prefix sum up the warp histograms
    if (threadIdx.x < RADIX)
    {
        uint32_t reduction = s_warpHistograms[threadIdx.x];
        for (uint32_t i = threadIdx.x + RADIX; i < BIN_HISTS_SIZE; i += RADIX)
        {
            reduction += s_warpHistograms[i];
            s_warpHistograms[i] = reduction - s_warpHistograms[i];
        }

        //begin the exclusive prefix sum across the reductions
        s_warpHistograms[threadIdx.x] = InclusiveWarpScanCircularShift(reduction);
    }
    __syncthreads();

    if (threadIdx.x < (RADIX >> LANE_LOG))
        s_warpHistograms[threadIdx.x << LANE_LOG] = ActiveExclusiveWarpScan(s_warpHistograms[threadIdx.x << LANE_LOG]);
    __syncthreads();

    if (threadIdx.x < RADIX && getLaneId())
        s_warpHistograms[threadIdx.x] += __shfl_sync(0xfffffffe, s_warpHistograms[threadIdx.x - 1], 1);
    __syncthreads();

    //update offsets
    if (WARP_INDEX)
    {
        #pragma unroll 
        for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
        {
            const uint32_t t2 = keys[i] >> radixShift & RADIX_MASK;
            offsets[i] += s_warpHist[t2] + s_warpHistograms[t2];
        }
    }
    else
    {
        #pragma unroll
        for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
            offsets[i] += s_warpHistograms[keys[i] >> radixShift & RADIX_MASK];
    }

    //load in threadblock reductions
    if (threadIdx.x < RADIX)
    {
        s_localHistogram[threadIdx.x] = globalHist[threadIdx.x + (radixShift << 5)] +
            passHist[threadIdx.x * gridDim.x + blockIdx.x] - s_warpHistograms[threadIdx.x];
    }
    __syncthreads();

    //scatter keys into shared memory
    #pragma unroll
    for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
        s_warpHistograms[offsets[i]] = keys[i];
    __syncthreads();

    //scatter runs of keys into device memory
    if (blockIdx.x < gridDim.x - 1)
    {
        #pragma unroll BIN_KEYS_PER_THREAD
        for (uint32_t i = threadIdx.x; i < BIN_PART_SIZE; i += blockDim.x)
            alt[s_localHistogram[s_warpHistograms[i] >> radixShift & RADIX_MASK] + i] = s_warpHistograms[i];
    }

    if (blockIdx.x == gridDim.x - 1)
    {
        const uint32_t finalPartSize = size - BIN_PART_START;
        for (uint32_t i = threadIdx.x; i < finalPartSize; i += blockDim.x)
            alt[s_localHistogram[s_warpHistograms[i] >> radixShift & RADIX_MASK] + i] = s_warpHistograms[i];
    }
}

__global__ void DeviceRadixSort::DownsweepPairs(
    uint32_t* sort,
    uint32_t* sortPayload,
    uint32_t* alt, 
    uint32_t* altPayload,
    uint32_t* globalHist,
    uint32_t* passHist,
    uint32_t size, 
    uint32_t radixShift)
{
    __shared__ uint32_t s_warpHistograms[BIN_PART_SIZE];
    __shared__ uint32_t s_localHistogram[RADIX];
    volatile uint32_t* s_warpHist = &s_warpHistograms[WARP_INDEX << RADIX_LOG];

    //clear shared memory
    for (uint32_t i = threadIdx.x; i < BIN_HISTS_SIZE; i += blockDim.x)
        s_warpHistograms[i] = 0;

    //load keys
    uint32_t keys[BIN_KEYS_PER_THREAD];
    if (blockIdx.x < gridDim.x - 1)
    {
        #pragma unroll
        for (uint32_t i = 0, t = getLaneId() + BIN_SUB_PART_START + BIN_PART_START; i < BIN_KEYS_PER_THREAD; ++i, t += LANE_COUNT)
            keys[i] = sort[t];
    }

    //To handle input sizes not perfect multiples of the partition tile size,
    //load "dummy" keys, which are keys with the highest possible digit.
    //Because of the stability of the sort, these keys are guaranteed to be 
    //last when scattered. This allows for effortless divergence free sorting
    //of the final partition.
    if (blockIdx.x == gridDim.x - 1)
    {
        #pragma unroll
        for (uint32_t i = 0, t = getLaneId() + BIN_SUB_PART_START + BIN_PART_START; i < BIN_KEYS_PER_THREAD; ++i, t += LANE_COUNT)
            keys[i] = t < size ? sort[t] : 0xffffffff;
    }
    __syncthreads();

    //WLMS
    uint16_t offsets[BIN_KEYS_PER_THREAD];
    #pragma unroll
    for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
    {
        unsigned warpFlags = 0xffffffff;
        #pragma unroll
        for (int k = 0; k < RADIX_LOG; ++k)
        {
            const bool t2 = keys[i] >> k + radixShift & 1;
            warpFlags &= (t2 ? 0 : 0xffffffff) ^ __ballot_sync(0xffffffff, t2);
        }
        const uint32_t bits = __popc(warpFlags & getLaneMaskLt());
        uint32_t preIncrementVal;
        if (bits == 0)
            preIncrementVal = atomicAdd((uint32_t*)&s_warpHist[keys[i] >> radixShift & RADIX_MASK], __popc(warpFlags));

        offsets[i] = __shfl_sync(0xffffffff, preIncrementVal, __ffs(warpFlags) - 1) + bits;
    }
    __syncthreads();

    //exclusive prefix sum up the warp histograms
    if (threadIdx.x < RADIX)
    {
        uint32_t reduction = s_warpHistograms[threadIdx.x];
        for (uint32_t i = threadIdx.x + RADIX; i < BIN_HISTS_SIZE; i += RADIX)
        {
            reduction += s_warpHistograms[i];
            s_warpHistograms[i] = reduction - s_warpHistograms[i];
        }

        //begin the exclusive prefix sum across the reductions
        s_warpHistograms[threadIdx.x] = InclusiveWarpScanCircularShift(reduction);
    }
    __syncthreads();

    if (threadIdx.x < (RADIX >> LANE_LOG))
        s_warpHistograms[threadIdx.x << LANE_LOG] = ActiveExclusiveWarpScan(s_warpHistograms[threadIdx.x << LANE_LOG]);
    __syncthreads();

    if (threadIdx.x < RADIX && getLaneId())
        s_warpHistograms[threadIdx.x] += __shfl_sync(0xfffffffe, s_warpHistograms[threadIdx.x - 1], 1);
    __syncthreads();

    //update offsets
    if (WARP_INDEX)
    {
        #pragma unroll 
        for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
        {
            const uint32_t t2 = keys[i] >> radixShift & RADIX_MASK;
            offsets[i] += s_warpHist[t2] + s_warpHistograms[t2];
        }
    }
    else
    {
        #pragma unroll
        for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
            offsets[i] += s_warpHistograms[keys[i] >> radixShift & RADIX_MASK];
    }

    //load in threadblock reductions
    if (threadIdx.x < RADIX)
    {
        s_localHistogram[threadIdx.x] = globalHist[threadIdx.x + (radixShift << 5)] +
            passHist[threadIdx.x * gridDim.x + blockIdx.x] - s_warpHistograms[threadIdx.x];
    }
    __syncthreads();

    //scatter keys into shared memory
    #pragma unroll
    for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
        s_warpHistograms[offsets[i]] = keys[i];
    __syncthreads();

    //scatter runs of keys into device memory
    uint8_t digits[BIN_KEYS_PER_THREAD];
    if (blockIdx.x < gridDim.x - 1)
    {
        //store the digit of key in register
        #pragma unroll
        for (uint32_t i = 0, t = threadIdx.x; i < BIN_KEYS_PER_THREAD;
            ++i, t += blockDim.x)
        {
            digits[i] = s_warpHistograms[t] >> radixShift & RADIX_MASK;
            alt[s_localHistogram[digits[i]] + t] = s_warpHistograms[t];
        }
        __syncthreads();

        //Load payloads into registers
        #pragma unroll
        for (uint32_t i = 0, t = getLaneId() + BIN_SUB_PART_START + BIN_PART_START;
            i < BIN_KEYS_PER_THREAD;
            ++i, t += LANE_COUNT)
        {
            keys[i] = sortPayload[t];
        }

        //scatter payloads into shared memory
        #pragma unroll
        for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
            s_warpHistograms[offsets[i]] = keys[i];
        __syncthreads();

        //Scatter the payloads into device
        #pragma unroll
        for (uint32_t i = 0, t = threadIdx.x; i < BIN_KEYS_PER_THREAD;
            ++i, t += blockDim.x)
        {
            altPayload[s_localHistogram[digits[i]] + t] = s_warpHistograms[t];
        }
    }

    if (blockIdx.x == gridDim.x - 1)
    {
        const uint32_t finalPartSize = size - BIN_PART_START;
        //store the digit of key in register
        #pragma unroll
        for (uint32_t i = 0, t = threadIdx.x; i < BIN_KEYS_PER_THREAD;
            ++i, t += blockDim.x)
        {
            if (t < finalPartSize)
            {
                digits[i] = s_warpHistograms[t] >> radixShift & RADIX_MASK;
                alt[s_localHistogram[digits[i]] + t] = s_warpHistograms[t];
            }
        }
        __syncthreads();

        //Load payloads into registers
        #pragma unroll
        for (uint32_t i = 0, t = getLaneId() + BIN_SUB_PART_START + BIN_PART_START;
            i < BIN_KEYS_PER_THREAD;
            ++i, t += LANE_COUNT)
        {
            if(t < size)
                keys[i] = sortPayload[t];
        }

        //scatter payloads into shared memory
        #pragma unroll
        for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i)
            s_warpHistograms[offsets[i]] = keys[i];
        __syncthreads();

        //Scatter the payloads into device
        #pragma unroll
        for (uint32_t i = 0, t = threadIdx.x; i < BIN_KEYS_PER_THREAD;
            ++i, t += blockDim.x)
        {
            if(t < finalPartSize)
                altPayload[s_localHistogram[digits[i]] + t] = s_warpHistograms[t];
        }
    }
}